
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
// #include <helper_cuda.h>
// #include <helper_string.h>

/* Run with only HOST code *\

// additional comment + some other additional comment
int main(void) {
	printf("Goodbye Universe!\n");
	// FUTURE: comment
	return 4;
}

*/


/* Run with DEVICE code */

__global__ void mykernel(void) { 
}

int main(void) {
  mykernel<<<1,1>>>();
  printf("Goodbye Universe!\n");
  // DEFECT: comment
  return 0;
}

/*
__global__  indicates function that 1) runs on DEVICE , 2) called from the HOST

<<<   >>>   indicates function that    runs on DEVICE and is called from the HOST

*/
