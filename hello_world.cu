
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
// #include <helper_cuda.h>
// #include <helper_string.h>

/* Run with only HOST code *\

// some other additional comment
int main(void) {
	printf("Hello Universe!\n");
	return 4;
}

*/


/* Run with DEVICE code */

__global__ void mykernel(void) { 
}

int main(void) {
  mykernel<<<1,1>>>();
  printf("Hello Universe!\n");
  // DEFECT: comment
  return 0;
}

/*
__global__  indicates function that 1) runs on DEVICE , 2) called from the HOST

<<<   >>>   indicates function that    runs on DEVICE and is called from the HOST

*/
