
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
// #include <helper_cuda.h>
// #include <helper_string.h>

/* Run with only HOST code *\

// additional comment
int main(void) {
	printf("Goodbye World!\n");
	// FUTURE: comment
	return 4;
}

*/


/* Run with DEVICE code */

__global__ void mykernel(void) { 
}

int main(void) {
  mykernel<<<1,1>>>();
  printf("Goodbye World!\n");
  return 0;
}

/*
__global__  indicates function that 1) runs on DEVICE , 2) called from the HOST

<<<   >>>   indicates function that    runs on DEVICE and is called from the HOST

*/
