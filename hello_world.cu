
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
// #include <helper_cuda.h>
// #include <helper_string.h>

/* Run with only HOST code *\

int main(void) {
	printf("Hello World!\n");
	return 4;
}

*/


/* Run with DEVICE code */

__global__ void mykernel(void) { 
}

int main(void) {
  mykernel<<<1,1>>>();
  printf("Hello World!\n");
  return 0;
}

/*
__global__  indicates function that 1) runs on DEVICE , 2) called from the HOST

<<<   >>>   indicates function that    runs on DEVICE and is called from the HOST

*/
