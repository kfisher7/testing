
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>

__global__ void add(int *a, int *b, int *c) {
  *c = *a + *b;
}

int main(void) {
  int a, b, c;          // HOST copies of a, b, c
  int *d_a, *d_b, *d_c;    // DEVICE copies of a, b, c
  
  int size = sizeof(int);

  // Allocate space on DEVICE for DEVICE copies d_a, d_b, d_c
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  // Set input values
  a = 2;
  b = 7;

  // Copy inputs to DEVICE
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on DEVICE
  add<<<1,1>>>(d_a, d_b, d_c);

  // Copy result back to HOST
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

  // Test
  if(c != a + b) std::cout << "Incorrect addition with result: " << c << '\n';

  //Clean-up 
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;

}

/* Moving to parallel

add<<<N, 1>>> we can execute add() N times in parallel

each parallel invocation is called a BLOCK
the collectino fo blocks is called a GRID

each invocation can refer to it's BLOCK index with
  blockIdx.x




*/